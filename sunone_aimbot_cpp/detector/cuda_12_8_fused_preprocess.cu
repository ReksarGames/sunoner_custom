#include "hip/hip_runtime.h"
﻿// cuda_12_8_fused_preprocess.cu
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>

// ---------------------------------------------
// Вспомогательная структура для letterbox
// ---------------------------------------------
struct Letterbox {
    int in_w, in_h;     // исходный размер
    int out_w, out_h;   // размер к которому приводим
    float scale;        // min(out_w/in_w, out_h/in_h)
    int pad_x;          // отступ слева
    int pad_y;          // отступ сверху
    float mean[3];      // mean по каналам (в [0..1])
    float std_[3];      // std по каналам (в [0..1])
};

// ---------------------------------------------
// Билинейная выборка из RGB uchar3
// src_pitch_pixels — шаг строки в пикселях (НЕ в байтах)
// ---------------------------------------------
__device__ __forceinline__ float3 sample_bilinear_rgb(
    const uchar3* __restrict__ src,
    int sw, int sh, int src_pitch_pixels,
    float x, float y)
{
    // clamp к валидным координатам
    x = fminf(fmaxf(x, 0.0f), (float)(sw - 1));
    y = fminf(fmaxf(y, 0.0f), (float)(sh - 1));

    int x0 = (int)floorf(x);
    int y0 = (int)floorf(y);
    int x1 = min(x0 + 1, sw - 1);
    int y1 = min(y0 + 1, sh - 1);

    float dx = x - x0;
    float dy = y - y0;

    const uchar3* row0 = src + y0 * src_pitch_pixels;
    const uchar3* row1 = src + y1 * src_pitch_pixels;

    uchar3 c00 = row0[x0];
    uchar3 c01 = row0[x1];
    uchar3 c10 = row1[x0];
    uchar3 c11 = row1[x1];

    // линейная интерполяция по X для двух строк
    float3 f0 = make_float3(
        c00.x + (c01.x - c00.x) * dx,
        c00.y + (c01.y - c00.y) * dx,
        c00.z + (c01.z - c00.z) * dx
    );
    float3 f1 = make_float3(
        c10.x + (c11.x - c10.x) * dx,
        c10.y + (c11.y - c10.y) * dx,
        c10.z + (c11.z - c10.z) * dx
    );

    // интерполяция по Y
    return make_float3(
        f0.x + (f1.x - f0.x) * dy,
        f0.y + (f1.y - f0.y) * dy,
        f0.z + (f1.z - f0.z) * dy
    );
}

// ---------------------------------------------
// Основное ядро: RGB8 -> NCHW float
// - Letterbox (масштаб + паддинг)
// - Билинейный ресайз
// - Нормализация (x-mean)/std
// ---------------------------------------------
__global__ void fused_preprocess_kernel_rgb8_to_nchw32f(
    const uchar3* __restrict__ src,
    int sw, int sh, int src_pitch_pixels,
    float* __restrict__ dst,
    Letterbox lb)
{
    int ox = blockIdx.x * blockDim.x + threadIdx.x; // x в выходе
    int oy = blockIdx.y * blockDim.y + threadIdx.y; // y в выходе
    if (ox >= lb.out_w || oy >= lb.out_h) return;

    // индекс пикселя в плоскости (H*W)
    int out_idx = oy * lb.out_w + ox;

    // координаты источника до паддинга
    float ix = (ox - lb.pad_x) / lb.scale;
    float iy = (oy - lb.pad_y) / lb.scale;

    float3 rgb;

    // если попали в зону паддинга — заливаем mean
    bool in_pad =
        (ox < lb.pad_x) ||
        (oy < lb.pad_y) ||
        (ox >= lb.pad_x + (int)(lb.in_w * lb.scale)) ||
        (oy >= lb.pad_y + (int)(lb.in_h * lb.scale));

    if (in_pad) {
        rgb = make_float3(lb.mean[0] * 255.0f, lb.mean[1] * 255.0f, lb.mean[2] * 255.0f);
    }
    else {
        rgb = sample_bilinear_rgb(src, sw, sh, src_pitch_pixels, ix, iy);
    }

    // в [0..1]
    rgb.x *= (1.0f / 255.0f);
    rgb.y *= (1.0f / 255.0f);
    rgb.z *= (1.0f / 255.0f);

    // нормализация
    rgb.x = (rgb.x - lb.mean[0]) / lb.std_[0];
    rgb.y = (rgb.y - lb.mean[1]) / lb.std_[1];
    rgb.z = (rgb.z - lb.mean[2]) / lb.std_[2];

    // запись в NCHW
    const int plane = lb.out_w * lb.out_h;
    dst[out_idx] = rgb.x;            // C0
    dst[plane + out_idx] = rgb.y;            // C1
    dst[plane * 2 + out_idx] = rgb.z;            // C2
}

// ---------------------------------------------
// Экспортируемый хелпер запуска (C-linkage!)
// mean/std — массивы из 3 элементов в [0..1]
// src_pitch_pixels — шаг строки в ПИКСЕЛЯХ (для uchar3)
// ---------------------------------------------
extern "C" void launch_fused_preprocess(
    const uchar3* d_rgb, int in_w, int in_h, int src_pitch_pixels,
    float* d_out, int out_w, int out_h,
    const float mean[3], const float std_[3],
    hipStream_t stream)
{
    Letterbox lb{};
    lb.in_w = in_w;   lb.in_h = in_h;
    lb.out_w = out_w;  lb.out_h = out_h;

    lb.scale = fminf((float)out_w / (float)in_w, (float)out_h / (float)in_h);
    lb.pad_x = (int)((out_w - in_w * lb.scale) * 0.5f);
    lb.pad_y = (int)((out_h - in_h * lb.scale) * 0.5f);

    lb.mean[0] = mean[0]; lb.mean[1] = mean[1]; lb.mean[2] = mean[2];
    lb.std_[0] = std_[0]; lb.std_[1] = std_[1]; lb.std_[2] = std_[2];

    dim3 block(32, 16);
    dim3 grid((out_w + block.x - 1) / block.x,
        (out_h + block.y - 1) / block.y);

    fused_preprocess_kernel_rgb8_to_nchw32f << <grid, block, 0, stream >> > (
        d_rgb, in_w, in_h, src_pitch_pixels,
        d_out, lb
        );

    // (необязательно) проверка запуска:
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) printf("fused_preprocess: %s\n", hipGetErrorString(err));
}
